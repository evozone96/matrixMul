#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void 
matrixMulCUDA(float *C, float *A, float *B, int wA, int wB)
{
    // Block index
//    int bx = blockIdx.x;
//    int by = blockIdx.y;

    // Thread index
//    int tx = threadIdx.x;
//    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
//    float aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
//    float aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
//    float aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
//    float  bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
//    float bStep  = BLOCK_SIZE * wB;
    float sum = 00.1;
    float fsum = 00.2;
    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
//    for (int a = aBegin, b = bBegin;
//         a <= aEnd;
//         a += aStep, b += bStep)
//    {
        

  
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
//        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
//        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
//        As[ty][tx] = A[a + wA * ty + tx];
//        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        //__syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
      int result;
        float qqq =0;
        float x_counter = 0.0;
        asm(".reg .f32 t1;\n\t");
        asm(".reg .u32 t2, t3, t4;\n\t");
#pragma unroll
        if (0) {
        for (int k = 0; k < BLOCK_SIZE; ++k) {
        //for (float k = 0.1; k < 32.9; k = k+0.99)
       //{
            while (x_counter < 1000000) {
            asm("mul.f32 %0, %3, t1, %2;\n\t"
                "add.u32 t2, t3, t4;\n\t"
                "mul.f32 t1, %0, t1, %3;\n\t"
                "mul.f32 t1, t1, t1, %2;\n\t"
                "add.u32 t2, t3, t4;\n\t"
                "mul.f32 t1, %0, t1, %0;\n\t"
                "mul.f32 %0, t1, t1, %0;\n\t"
                "mul.f32 t1, %0, t1, %0;\n\t"
                "mul.f32 t1, t1, t1, %2;\n\t"
                "mul.f32 t1, %0, t1, %0;\n\t"
                "add.u32 t4, t3, t2;\n\t"
                "mul.f32 %0, t1, %0, %3;\n\t"
                "mul.f32 t1, %0, %3, %0;\n\t"
                "mul.f32 t1, t1, %2, %0;\n\t"
                "mul.f32 t1, %0, %0, %3;\n\t"
                "add.u32 t2, t2, t4;\n\t"
                "mul.f32 %0, t1, %0, t1;\n\t"
                "mul.f32 t1, t1, %0, %0;\n\t"
                "add.u32 %1, t2, t4;\n\t"
                "mul.f32 %0, t1, %0, t1;\n\t": "=f"(qqq), "=r"(result): "f"(sum), "f"(fsum) );

                x_counter += 1.0;
           // }
            //qqq += k*k;
            //sum += qqq*qqq/(qqq*2.3);
            //sum += (a+b+k)*qqq;
            //Csub += As[ty][k] * Bs[k][tx] + sum;
        }
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        //__syncthreads();
    }

    if (1) {
    //if (threadIdx.y % 2 == 0) {

        //for (int k = 0; k < BLOCK_SIZE; ++k) {
        //for (float k = 0.1; k < 32.9; k = k+0.99)
       //{
            while (x_counter < 10000000) {
            asm("add.u32 t2, t3, t4;\n\t"
                "mul.f32 %0, t1, %0;\n\t"
                "add.u32 t2, t2, t4;\n\t"
                "mul.f32 t1, t1, %0;\n\t"
                "add.u32 t2, t2, t4;\n\t"
                "mul.f32 t1, t1, %0;\n\t"
                "add.u32 t4, t3, t2;\n\t"
                "mul.f32 %0, t1, %0;\n\t"
                "add.u32 t2, t2, t4;\n\t"
                "mul.f32 t1, t1, %2;\n\t": "=f"(qqq), "=r"(result): "f"(sum), "f"(fsum) );

                x_counter += 1.0;
            }
        //}
    }

    /*
    if (threadIdx.y % 2 == 0) {

        //for (int k = 0; k < BLOCK_SIZE; ++k) {
        //for (float k = 0.1; k < 32.9; k = k+0.99)
       //{
            while (x_counter < 10000000) {
            asm("add.u32 t2, t3, t4;\n\t"
                "add.u32 t4, t3, t2;\n\t"
                "add.u32 t2, t2, t4;\n\t"
                "add.u32 t3, t3, t2;\n\t"
                "add.u32 t2, t2, t4;\n\t"
                "add.u32 t2, t3, t4;\n\t"
                "add.u32 t4, t3, t2;\n\t"
                "add.u32 t2, t2, t4;\n\t"
                "add.u32 t4, t3, t2;\n\t"
                "add.u32 t4, t3, t2;\n\t": "=f"(qqq), "=r"(result): "f"(sum), "f"(fsum) );

                x_counter += 1.0;
            }
        //}
    }else {

        //for (int k = 0; k < BLOCK_SIZE; ++k) {
        //for (float k = 0.1; k < 32.9; k = k+0.99)
       //{
            while (x_counter < 10000000) {
            asm("mul.f32 t1, %0, t1;\n\t"
                "mul.f32 %0, t1, %0;\n\t"
                "mul.f32 t1, %0, %0;\n\t"
                "mul.f32 %2, t1, %0;\n\t"
                "mul.f32 %2, t1, %0;\n\t"
                "mul.f32 %0, t1, %0;\n\t"
                "mul.f32 t1, t1, %2;\n\t"
                "mul.f32 %2, t1, %0;\n\t"
                "mul.f32 %0, t1, %0;\n\t"
                "mul.f32 t1, t1, %2;\n\t": "=f"(qqq), "=r"(result): "f"(sum), "f"(fsum) );

                x_counter += 1.0;
            }
        //}
    }
    */
    // Write the block sub-matrix to device memory;
    // each thread writes one element
    //int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    //C[c + wB * ty + tx] = Csub;
    C[0] = qqq;
}

void constantInit(float *data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}


/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void
matrixMulCUDA_int(int *C, int *A, int *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;
    int sum = 0;
    int fsum = 0;
    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    int Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {
        

  
        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ int As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        //__syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll
        int qqq =0;
        int x_counter = 0;
        asm(".reg .u32 t1;\n\t");
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            while (x_counter < 1000000) {
            asm("mul.u32 %0, %1, %2;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, %0, %1;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 t1, t1, %2;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, t1, %0;\n\t"
                "mul.u32 t1, %0, %0;\n\t"
                "mul.u32 %0, %0, t1;\n\t": "=r"(qqq): "r"(As[ty][k]), "r"(Bs[k][tx]) );

                x_counter += 1;
            }

            //qqq += k*k;
            //fsum += qqq*qqq/(qqq*3);
            //sum += a+b+k;
            //Csub += As[ty][k] * Bs[k][tx]+sum;
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        //__syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    //int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    //C[c + wB * ty + tx] = Csub;
}

void constantInit_int(int *data, int size, int val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int matrixMultiply(int argc, char **argv, int block_size, dim3 &dimsA, dim3 &dimsB)
{
    int streamNum = 1;
    if (checkCmdLineFlag(argc, (const char **)argv, "streams"))
    {
        streamNum = getCmdLineArgumentInt(argc, (const char **)argv, "streams");
    }
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    unsigned int mem_size_A_double = sizeof(int) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    unsigned int mem_size_B_double = sizeof(int) * size_B;
    float *h_B = (float *)malloc(mem_size_B);
    int *h_A_double = (int *)malloc(mem_size_A_double);
    int *h_B_double = (int *)malloc(mem_size_B_double);

    // Initialize host memory
    const float valB = 0.01f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);
    constantInit_int(h_A_double, size_A, 2);
    constantInit_int(h_B_double, size_B, 23);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    // Allocate device memory
    int *d_A_double, *d_B_double, *d_C_double;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);
    unsigned int mem_size_C_double = dimsC.x * dimsC.y * sizeof(int);
    int *h_C_double = (int *) malloc(mem_size_C_double);


    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(streamNum * sizeof(hipStream_t));

    for (int i = 0; i < streamNum; i++)
    {
        checkCudaErrors(hipStreamCreate(&(streams[i])));
    }
    if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    hipError_t error;

    error = hipMalloc((void **) &d_A, mem_size_A);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_B, mem_size_B);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_C, mem_size_C);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_A_double, mem_size_A_double);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_B_double, mem_size_B_double);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_C_double, mem_size_C_double);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    error = hipMemcpy(d_A_double, h_A_double, mem_size_A_double, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B_double, h_B_double, mem_size_B_double, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }


    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);


    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16)
    {
//        matrixMulCUDA<16><<< grid, threads, 0,streams[0] >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
    else
    {
//        matrixMulCUDA<32><<< grid, threads, 0, streams[0] >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }

    printf("done\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }


    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Execute the kernel
    int nIter = 4;

    for (int j = 0; j < nIter; j++)
    {
        if (block_size == 16)
        {
            matrixMulCUDA<16><<< grid, threads,0, streams[j%streamNum] >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else
        {
            matrixMulCUDA<32><<< grid, threads,0, streams[j%streamNum] >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        if (block_size == 16)
        {
            matrixMulCUDA_int<16><<< grid, threads,0, streams[(j+1)%streamNum] >>>(d_C_double, d_A_double, d_B_double, dimsA.x, dimsB.x);
        }
        else
        {
            matrixMulCUDA_int<32><<< grid, threads,0, streams[(j+1)%streamNum] >>>(d_C_double, d_A_double, d_B_double, dimsA.x, dimsB.x);
        }
    }
    // Record the start event
    error = hipEventRecord(start, NULL);

    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6 ; // machine zero

    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
    {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err/abs_val/dot_length ;

        if (rel_err > eps)
        {
       //     printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x*valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A_double);
    free(h_B_double);
    free(h_C_double);
    hipFree(d_A_double);
    hipFree(d_B_double);
    hipFree(d_C_double);

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}


/**
 * Program main
 */
int main(int argc, char **argv)
{
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;
    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    if ((deviceProp.concurrentKernels == 0))
    {
        printf("> GPU does not support concurrent kernel execution\n");
        printf("  CUDA kernel runs will be serialized\n");
    }

    printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
    int block_size = 32;

    dim3 dimsA(5*2*block_size, 5*2*block_size, 1);
    dim3 dimsB(5*4*block_size, 5*2*block_size, 1);

    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA"))
    {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA"))
    {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB"))
    {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB"))
    {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y)
    {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

    int matrix_result = matrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}
